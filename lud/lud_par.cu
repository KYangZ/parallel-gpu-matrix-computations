#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include<sys/time.h>
#include "io.h"
#include "seq.cpp"

#define HANDLE_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void print_matrix(float* mat, int N){
    printf("[");
    for(int i=0;i< N*N;i++){
        printf("%.2f\t",mat[i]);
        if((i+1)%N==0){
            printf("\n");
        }
    }
    printf("]\n");
}

void delete_matrix(float* mat, int N){
    free(mat);
}
void gen_matrix(float* mat, int N){
    for (int i=0;i<N*N;i++){
		mat[i] = ((rand()%10)+1);
	}
}

__global__ void scale(float *g_odata, float *mat, int size, int row_index){
   int thid = threadIdx.x + blockIdx.x * blockDim.x;
   if (thid<size-row_index){
      g_odata[row_index*size+(thid)*size+row_index] = 
      mat[row_index*size+(thid)*size+row_index]/mat[row_index*size+row_index];
   }
}
__global__ void reduce(float *g_idata, float *mat, int size, int row_index){
   int thid = threadIdx.x + blockIdx.x * blockDim.x;
   if (thid<(size - row_index-1) * (size - row_index)){
      int row = thid/(size - row_index);
      int col = thid%(size - row_index);
      int i = row_index*size + ((row+1)*size)+(col+row_index);
      mat[i] = mat[i] - g_idata[row_index*size + 
      ((row+1)*size)+row_index]*mat[(row_index*size)+col+row_index];
   }
}
//Given square matrix in flattened format with dim, compute LU decomp in parallel
void lud_parallel(float* l, float* u, float* matrix, int N){
    float *dev_matrix, *dev_l;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    HANDLE_ERROR(hipMalloc((void**)&dev_matrix, N*N*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_l, N*N*sizeof(float)));
    HANDLE_ERROR(hipMemcpy(dev_matrix, matrix, N*N*sizeof(float), hipMemcpyHostToDevice));

    //start timer
    hipEventRecord(start);

    for(int i=0;i<N;i++){
        int n = N-i;
        scale<<<(n+127)/128, 128>>>(dev_l, dev_matrix, N, i);
        reduce<<<((n*n)+127)/128, 128>>>(dev_l, dev_matrix, N, i);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Parallel runtime for %dx%d dimensions: %f\n",N,N,milliseconds);

    HANDLE_ERROR(hipMemcpy(l, dev_l, N*N*sizeof(float), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(u, dev_matrix, N*N*sizeof(float), hipMemcpyDeviceToHost));
    hipFree(dev_l);
    hipFree(dev_matrix);
}

void simple_verification(int N, float* mat){
    print_matrix(mat, N);

    float *l = new float[N*N];
    float *u = new float[N*N];

    lud_parallel(l, u, mat, N);

    print_matrix(l, N);
    print_matrix(u, N);

    free(l);
    free(u);
    free(mat);
}
void copy_matrix(float* mat1, float* mat2, int N){
    for(int i=0;i<N*N;i++){
        mat1[i] = mat2[i];
    }
}
void compare_n_sizes(){
    int Nums[6] = {25, 100, 500, 1000, 2500, 5000};
    int N;
    for(int i=0;i<6;i++){
        N = Nums[i];
        printf("Comparing N dim: %d by %d \n", N, N);

        float *mat = new float[N*N];
        gen_matrix(mat, N);
        float *l = new float[N*N];
        float *u = new float[N*N];

        lud_parallel(l, u, mat, N);
        
        float** l_par_grid = new float*[N];
        float** u_par_grid = new float*[N];

        create_matrix_seq(l_par_grid, N);
        create_matrix_seq(u_par_grid, N);

        flat_to_grid(l_par_grid, l, N);
        flat_to_grid(u_par_grid, u, N);
        free(l);
        free(u);

        float** l_seq = new float*[N];
        float** u_seq = new float*[N];
        create_matrix_seq(l_seq, N);
        create_matrix_seq(u_seq, N);
        flat_to_grid(u_seq, mat, N);
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

            //start timer
        hipEventRecord(start);
        lud_sequential(l_seq, u_seq, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Sequential runtime for %dx%d dimensions: %f\n",N,N,milliseconds);
  
        delete(mat);


    }
}
int main(int argc, char** argv){

    int N = atoi(argv[1]);
    srand(42);
    float *mat = new float[N*N];
    float *mat_copy = new float[N*N];
    gen_matrix(mat, N);
    copy_matrix(mat_copy, mat, N);
    printf("Parallel implementation: \n");
    simple_verification(N, mat);
    printf("Sequential implementation: \n");
    simple_verification_seq(N, mat_copy);
    free(mat_copy);
    free(mat);
    // printf("Sweeping over multiple sizes...\n");
    // compare_n_sizes();
    return 0;
}

