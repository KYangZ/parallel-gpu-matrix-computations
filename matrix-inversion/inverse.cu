#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>

#pragma comment(lib, "cuda.lib")
#pragma comment(lib, "cudart.lib")
#include <hip/hip_runtime.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

#include <hipblas.h>

using namespace std;

#define blocksize 8

// normalize elements not in diagonal
__global__ void nodiag_normalize(double *A, double *I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x == i && x!=y) {
			I[x*n + y] /= A[i*n + i];
			A[x*n + y] /= A[i*n + i];
		}
	}
}

// normalize elements in diagonal
__global__ void diag_normalize(double *A, double *I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x == y && x == i) {
			I[x*n + y] /= A[i*n + i];
			A[x*n + y] /= A[i*n + i];
		}
	}

}

// realization of parallel matrix inverse 
__global__ void gaussjordan(double *A, double *I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x != i) {
			I[x*n + y] -= I[i*n + y] * A[x*n + i];
			if (y != i) {
				A[x*n + y] -= A[i*n + y] * A[x*n + i];
			}	 
		}
	}

}

// set first line in every loop to 0
__global__ void set_zero(double *A, double *I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x != i) {
			if (y == i) {
				A[x*n + y] = 0;
			}
		}
	}
}


/** start of local helper functions **/
void matrix_read(char filename[], double *L, int dimension) {
	FILE *fp;
	int row, col;

	fp = fopen(filename, "r");//open output file
	if (fp == NULL)//open failed
		return;

	for (row = 0; row < dimension; row++) {
		for (col = 0; col < dimension; col++)
		if (fscanf(fp, "%f,", &L[row * dimension + col]) == EOF) break;//read data

		if (feof(fp)) break;//if the file is over
	}

	fclose(fp);//close file
}

void savetofile(double *A, string s, int n, int h) {
	std::ofstream plik;
	plik.open(s);

	for (int j = 0; j<h; j++) {
		for (int i = 0; i<h; i++) {
			plik << A[j*n + i] << "\t";
		}
		plik << endl;
	}
	plik.close();
}
/** end of local helper functions **/

void invert(char filename[], int n) {
	// creating input
	double *iL = new double[n*n];
	double *L = new double[n*n];
	matrix_read(filename, L, n);

	// initialization 
	double *d_A, *I, *dI;
	float time;
	int ddsize = n*n*sizeof(double);

	// memory allocation    
	dim3 threadsPerBlock(blocksize, blocksize);
	dim3 numBlocks((n + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);

	hipMalloc((void**)&d_A, ddsize);
	hipMalloc((void**)&dI, ddsize);
	
	// identity matrix
	I = new double[n*n];
	for (int i = 0; i<n; i++) {
		for (int j = 0; j<n; j++) {
			if (i == j) I[i*n + i] = 1.0;
			else I[i*n + j] = 0.0;
		}
	}

	//copy data from CPU to GPU
	hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice);
	hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);

	//timer start
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// L^(-1)    
	for (int i = 0; i<n; i++) {
		nodiag_normalize <<< numBlocks, threadsPerBlock >>>(d_A, dI, n, i);
		diag_normalize <<< numBlocks, threadsPerBlock >>>(d_A, dI, n, i);
		gaussjordan <<< numBlocks, threadsPerBlock >>>(d_A, dI, n, i);
		set_zero <<< numBlocks, threadsPerBlock >>>(d_A, dI, n, i);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//copy data from GPU to CPU
	hipMemcpy(iL, dI, ddsize, hipMemcpyDeviceToHost);
	hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);
	
	
	savetofile(iL, "outputs/inv.txt", n, n);

	// double *c = new double[n*n];
	// for (int i = 0; i<n; i++)  {}
	// 	for (int j = 0; j<n; j++) {
	// 		c[i*n+j] = 0;  //put the initial value to zero

	// 		for (int x = 0; x<n; x++) {
	// 			c[i*n + j] = c[i*n + j] + L[i*n+x] * iL[x*n + j];  //matrix multiplication
	// 		}
	// 	}
	// }
	// savetofile(c, "outputs/c.txt", n, n);

	// free variables 
	hipFree(d_A);
	hipFree(dI);
	delete[]I;
	delete[]L;
	delete[]iL;
}

int main(int argc, char *argv[]) {
	char* file;
	int n;

	file = argv[1];
	n = stoi(argv[2]);

	invert(file, n);
    return 0;
}
